#include "hip/hip_runtime.h"
#include "solver.hpp"
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>
#include <>
#include <iostream>

__constant__ float c_conductivities[MAX_MATERIALS];
__constant__ float c_densities[MAX_MATERIALS];
__constant__ float c_specific_heats[MAX_MATERIALS];

__global__ void heatStepKernel(
    float* out,
    const float* in,
    const int* material_ids,
    int width,
    int height,
    float dt
) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < 1 || x >= width - 1 || y < 1 || y >= height - 1) {
        if (x < width && y < height) {
            int idx = y * width + x;
            out[idx] = in[idx];  
        }
        return;
    }

    int idx = y * width + x;
    int mat_id = material_ids[idx];

    if (mat_id < 0) {
        out[idx] = in[idx];
        return;
    }

    float k_center = c_conductivities[mat_id];
    float rho = c_densities[mat_id];
    float c = c_specific_heats[mat_id];

    int id_left = material_ids[y * width + (x - 1)];
    int id_right = material_ids[y * width + (x + 1)];
    int id_up = material_ids[(y - 1) * width + x];
    int id_down = material_ids[(y + 1) * width + x];

    float k_left = (id_left >= 0) ?
        2.0f * k_center * c_conductivities[id_left] / (k_center + c_conductivities[id_left]) : k_center;
    float k_right = (id_right >= 0) ?
        2.0f * k_center * c_conductivities[id_right] / (k_center + c_conductivities[id_right]) : k_center;
    float k_up = (id_up >= 0) ?
        2.0f * k_center * c_conductivities[id_up] / (k_center + c_conductivities[id_up]) : k_center;
    float k_down = (id_down >= 0) ?
        2.0f * k_center * c_conductivities[id_down] / (k_center + c_conductivities[id_down]) : k_center;

    float temp_center = in[idx];
    float temp_left = in[y * width + (x - 1)];
    float temp_right = in[y * width + (x + 1)];
    float temp_up = in[(y - 1) * width + x];
    float temp_down = in[(y + 1) * width + x];

    float flux = k_left * (temp_left - temp_center) +
        k_right * (temp_right - temp_center) +
        k_up * (temp_up - temp_center) +
        k_down * (temp_down - temp_center);

    out[idx] = temp_center + (dt * flux) / (rho * c);
}

GPUSolver::GPUSolver(const Config& config, const Grid& grid)
    : width_(grid.width()), height_(grid.height()),
    d_temp_in_(nullptr), d_temp_out_(nullptr),
    d_material_ids_(nullptr)
{
    size_t grid_size = width_ * height_;
    size_t temp_bytes = grid_size * sizeof(float);
    size_t id_bytes = grid_size * sizeof(int);

    hipMalloc(&d_temp_in_, temp_bytes);
    hipMalloc(&d_temp_out_, temp_bytes);
    hipMalloc(&d_material_ids_, id_bytes);

    const auto& materials = config.materials();
    size_t mat_count = materials.size();

    if (mat_count > MAX_MATERIALS) {
        throw std::runtime_error("Too many materials! Increase MAX_MATERIALS");
    }

    std::vector<float> conductivities(MAX_MATERIALS, 0.0f);
    std::vector<float> densities(MAX_MATERIALS, 0.0f);
    std::vector<float> specific_heats(MAX_MATERIALS, 0.0f);

    for (size_t i = 0; i < mat_count; ++i) {
        conductivities[i] = materials[i].thermal_conductivity;
        densities[i] = materials[i].density;
        specific_heats[i] = materials[i].specific_heat;
    }

    hipMemcpyToSymbol(HIP_SYMBOL(c_conductivities), conductivities.data(), MAX_MATERIALS * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_densities), densities.data(), MAX_MATERIALS * sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(c_specific_heats), specific_heats.data(), MAX_MATERIALS * sizeof(float));

    hipMemcpy(d_material_ids_, grid.material_ids(), id_bytes, hipMemcpyHostToDevice);
}

GPUSolver::~GPUSolver() {
    hipFree(d_temp_in_);
    hipFree(d_temp_out_);
    hipFree(d_material_ids_);
}

void GPUSolver::step(float* temp, float dt) {
    size_t grid_size = width_ * height_;
    size_t temp_bytes = grid_size * sizeof(float);

    hipMemcpy(d_temp_in_, temp, temp_bytes, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width_ + 15) / 16, (height_ + 15) / 16);

    heatStepKernel << <gridDim, blockDim >> > (
        d_temp_out_,
        d_temp_in_,
        d_material_ids_,
        width_,
        height_,
        dt
        );

    hipMemcpy(temp, d_temp_out_, temp_bytes, hipMemcpyDeviceToHost);
}

void GPUSolver::sync() {
    hipDeviceSynchronize();
}